#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
//#include <complex>

#include "cudacomplex.h"

template < typename T >
__global__ void kernel( T v )
{
    v = v + v;
}

int main()
{
    int N = 1000;
    dim3  dimBlock(64, 1, 1);
    dim3  dimGrid((N + dimBlock.x - 1)/dimBlock.x, 1, 1);

    //singlecomplex v1 = cplx::one();
    doublecomplex v1 = doublecomplex::one();
    //double v1 = 2;

    kernel<<<dimGrid , dimBlock>>>( v1 );
}
